#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

/* 
 * File:   main.cpp
 * Author: alterith
 *
 * Created on 05 May 2018, 3:40 PM
 */

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <cmath>
#include <vector>
#include "octree.h"
#include "body.h"
#include "arr_node.h"

#define THETA 0.5
#define G 6.674*(pow(10,-11))
#define delta_t 0.1

using namespace std;


/*
 * function definitions
 */

void update_point(arr_node* arr_tree, int* h_body_pos, vector<body *> *point, int num_ele);
int get_points(int n, node* octree_node);
int convert_tree_to_array(int n, node* octree_node, arr_node* arr_tree, int* h_body_pos, int* pos);

/*
 *
 */


/*
 * device definitions
 */
__device__ double distance_com(arr_node a, arr_node b) {
    double distance = (double)sqrt((double)pow((a.com_x - b.com_x), 2) + (double)pow((a.com_y - b.com_y), 2) + (double)pow((a.com_z - b.com_z), 2));
    return distance;
}

__device__ double size_region(arr_node a) {
    //perimeter
    double size = pow((a.max_x - a.min_x), 2);
    return size;
}

__global__ void body_update_kernel(int n, int num_ele, arr_node* arr_tree, int* pos){
    //obtain index
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    arr_node* singular_body;
    if(idx<num_ele){
        //use direct access to change things on source
        singular_body = &arr_tree[pos[idx]];
        //printf("Device: %d\t%d\t%d\n", pos[idx], singular_body->furthest_right, singular_body->body_num);
        //TODO: do traversal and force update calculation
        for(int i = 0; i<n; i++){
            //skip body if its the same point, pos is a singular body index in the arr_tree array
            if(i != pos[idx]){
                //calculate distance
                double dist = distance_com(*singular_body, arr_tree[i]);
                //if singular body no need to check if well seperated
                if(arr_tree[i].body_num != -1){
                    // newtons law of universal gravitation
                    double force_x = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_x - arr_tree[i].com_x);
                    double force_y = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_y - arr_tree[i].com_y);
                    double force_z = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_z - arr_tree[i].com_z);
                    singular_body->force_x += force_x;
                    singular_body->force_y += force_y;
                    singular_body->force_z += force_z;
                    //printf("Distance: %lf\n", dist);
                }else{
                    
                    //obtain perimeter of octant
                    double perimeter = size_region(arr_tree[i]);
                    //check if well separated
                    //printf("%lf\t%lf\t%lf\n", perimeter/dist, perimeter, dist);
                    if ((perimeter / dist) <= THETA) {
                        // we may treat the nodes in this octant as one and proceed
                        //newtons law of universal gravitation: (G*m1*m2/r^3)*r_(x,y,z)
                        double force_x = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_x - arr_tree[i].com_x);
                        double force_y = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_y - arr_tree[i].com_y);
                        double force_z = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3))*(singular_body->com_z - arr_tree[i].com_z);
                        singular_body->force_x += force_x;
                        singular_body->force_y += force_y;
                        singular_body->force_z += force_z;
                        //skip children
                        //printf("skipped\n");
                        i = arr_tree[i].furthest_right;
                    }
                }        
            }
        }
        __syncthreads();
        //TODO: do point update
        //do velocity calc change here
        singular_body->vel_x = singular_body->vel_x + (singular_body->force_x/singular_body->mass)*delta_t;
        singular_body->vel_y = singular_body->vel_y + (singular_body->force_y/singular_body->mass)*delta_t;
        singular_body->vel_z = singular_body->vel_z + (singular_body->force_z/singular_body->mass)*delta_t;
        //update position
        singular_body->com_x = singular_body->com_x + singular_body->vel_x*delta_t;
        singular_body->com_y = singular_body->com_y + singular_body->vel_y*delta_t;
        singular_body->com_z = singular_body->com_z + singular_body->vel_z*delta_t;
    }
}
 
 /*
  *
  */

int main(int argc, char **argv)
{

    int i = 0;
    int bound = 8;
    const int num_ele = 4000;
    int iterations = 100;
    srand(time(NULL)); //just seed the generator

    //mass, pos x,y,z, vel x,y,z
    //double point[num_ele][7];
    vector<body *> *point = new vector<body *>(num_ele);
    //create point array
    for (i = 0; i < num_ele; i++)
    {
        //mass
        double mass = (float)((rand() % 20000) / 1000.0) + 1.0;

        //position
        dim3float pos(
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound,
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound,
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound);

        //velocity
        dim3float vel(
            (float)((rand() % 40000) - 20000) / 1000.0,
            (float)((rand() % 40000) - 20000) / 1000.0,
            (float)((rand() % 40000) - 20000) / 1000.0);

        //insert
        body *newBody = create_body(mass, pos, vel);
        (*point)[i] = (newBody);
    }
    //insert into tree and run
    for (int i = 0; i < iterations; i++)
    {
        node *test = malloc_node(-1 * bound, -1 * bound, -1 * bound, bound, bound, bound);
        for (int i = 0; i < num_ele; i++)
        {
            int ele = insert_node(test, (*point)[i], i);
        }

        //just some testing for array representation;
        int n = get_points(0, test);
        //cout<<n<<endl;
        //declare pointers to host and device memory
        arr_node* h_arr_tree;
        arr_node* d_arr_tree;
        //individual body positions in array
        int* h_body_pos;
        int* d_body_pos;
        //allocate host memory
        h_arr_tree = (arr_node*)malloc(n*sizeof(arr_node));
        h_body_pos = (int*)malloc(num_ele*sizeof(int));
        //populate host array
        int pos = 0;
        int garbage = convert_tree_to_array(0, test, h_arr_tree, h_body_pos, &pos);
        //free the tree
        free_node(test);
        /*
        for(int k = 0; k < n; k++){
            printf("Host: %d\t%d\t%d\n", k, h_arr_tree[k].furthest_right, h_arr_tree[k].body_num);
        }
        cout<<endl;
        */
        //device memory allocation
        hipMalloc((void **)&d_arr_tree, n*sizeof(arr_node));
        hipMalloc((void **)&d_body_pos, num_ele*sizeof(int));
        
        int nblocks  = 3;
        int nthreads = num_ele/nblocks;
        //check if enough threads exist
        if(nthreads*nblocks != num_ele){
            nthreads++;
        }

        //copy relevant data to device
        hipMemcpy(d_arr_tree, h_arr_tree, n*sizeof(arr_node), hipMemcpyHostToDevice);
        hipMemcpy(d_body_pos, h_body_pos, num_ele*sizeof(int), hipMemcpyHostToDevice);
        
        body_update_kernel<<<nblocks,nthreads>>>(n, num_ele, d_arr_tree, d_body_pos);
        hipDeviceSynchronize();
        hipMemcpy(h_arr_tree, d_arr_tree, n*sizeof(arr_node), hipMemcpyDeviceToHost);
        //end of array representation testing

        //insert function updating the bodies
        update_point(h_arr_tree, h_body_pos, point, num_ele);

        //free memory
        free(h_arr_tree);
        free(h_body_pos);
        hipFree(d_arr_tree);
        hipFree(d_body_pos);
    }/*
    for(int i = 0; i<100;i++){
        cout<<(((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound)<<endl;
    }*/
    return 0;
}

//to start pass num_ele = 0
void update_point(arr_node* arr_tree, int* h_body_pos, vector<body *> *point, int num_ele)
{
    for(int i = 0; i < num_ele; i++){
        //get singular body
        arr_node temp = arr_tree[h_body_pos[i]];
        //get vector index
        int idx = temp.body_num;
        //allocate updated values
        (*point)[idx]->mass = temp.mass;
        (*point)[idx]->com.x = temp.com_x;
        (*point)[idx]->com.y = temp.com_y;
        (*point)[idx]->com.z = temp.com_z;
        (*point)[idx]->vel.x = temp.vel_x;
        (*point)[idx]->vel.y = temp.vel_y;
        (*point)[idx]->vel.z = temp.vel_z;
    }
}

// get num points in tree

int get_points(int n, node* octree_node){

    n +=1;
    for (int i = 0; i < 8; i++)
    {
        if (octree_node->children[i])
        {
            n = get_points(n, octree_node->children[i]);
        }
    }
    return n;

}

int convert_tree_to_array(int n, node* octree_node, arr_node* arr_tree, int* h_body_pos, int* pos){
    //store relevant data into array from node
    arr_tree[n].body_num = octree_node->body_num;
    arr_tree[n].com_x = octree_node->com.x;
    arr_tree[n].com_y = octree_node->com.y;
    arr_tree[n].com_z = octree_node->com.z;
    arr_tree[n].max_x = octree_node->max.x;
    arr_tree[n].max_y = octree_node->max.y;
    arr_tree[n].max_z = octree_node->max.z;
    arr_tree[n].min_x = octree_node->min.x;
    arr_tree[n].min_y = octree_node->min.y;
    arr_tree[n].min_z = octree_node->min.z;
    arr_tree[n].mass = octree_node->mass;
    arr_tree[n].vel_x = octree_node->vel.x;
    arr_tree[n].vel_y = octree_node->vel.y;
    arr_tree[n].vel_z = octree_node->vel.z;
    if(arr_tree[n].body_num != -1){
        h_body_pos[*pos] = n;
        *pos = *pos + 1;
    }
    int n1 = n;
    n +=1;
    //traverse children recursively
    for (int i = 0; i < 8; i++)
    {
        if (octree_node->children[i])
        {
            n = convert_tree_to_array(n, octree_node->children[i], arr_tree, h_body_pos, pos);
        }
    }
    //store furthest right node for skipping
    arr_tree[n1].furthest_right = n-1;
    return n;
}