#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

/*
 * File:   main.cpp
 * Author: alterith
 *
 * Created on 05 May 2018, 3:40 PM
 */

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <cmath>
#include <vector>
#include "octree.h"
#include "body.h"
#include "arr_node.h"
// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define THETA 0.5
#define G -6.674*(pow(10,-11))
#define delta_t 0.1

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;

const int writeOut = 1;


/*
 * function definitions
 */

void update_point(arr_node* arr_tree, int* h_body_pos, vector<body *> *point, int num_ele);
int get_points(int n, node* octree_node);
int convert_tree_to_array(int n, node* octree_node, arr_node* arr_tree, int* h_body_pos, int* pos);

/*
 *
 */


/*
 * device definitions
 */
__device__ double distance_com(arr_node a, arr_node b) {
    double distance = (double)sqrt((double)pow((a.com_x - b.com_x), 2) + (double)pow((a.com_y - b.com_y), 2) + (double)pow((a.com_z - b.com_z), 2));
    return distance;
}

__device__ double size_region(arr_node a) {
    //perimeter
    double size = pow((a.max_x - a.min_x), 2);
    return size;
}

__global__ void body_update_kernel(int n, int num_ele, arr_node* arr_tree, int* pos, int bound){
    //obtain index
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    arr_node* singular_body;
    if(idx<num_ele){
        //use direct access to change things on source
        singular_body = &arr_tree[pos[idx]];
        //printf("Device: %d\t%d\t%d\n", pos[idx], singular_body->furthest_right, singular_body->body_num);
        //TODO: do traversal and force update calculation
        for(int i = 0; i<n; i++){
            //skip body if its the same point, pos is a singular body index in the arr_tree array
            if(i != pos[idx]){
                //calculate distance
                double dist = distance_com(*singular_body, arr_tree[i]);
                //if singular body no need to check if well seperated
                if(arr_tree[i].body_num != -1){
                    // newtons law of universal gravitation
                    double force_x = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_x - arr_tree[i].com_x);
                    double force_y = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_y - arr_tree[i].com_y);
                    double force_z = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_z - arr_tree[i].com_z);
					                  
					//printf("%.11lf\n", force_x);
                    //10.10
                    if((force_x!=force_x)){
                      force_x = 1000;
                      //printf("force_x\n");
                    }
                    if((force_y!=force_y)){
                      force_y = 1000;
                      //printf("force_y\n");
                    }
                    if((force_z!=force_z)){
                      force_z = 1000;
                      //printf("force_z\n");
                    }

					
                    singular_body->force_x += force_x;
                    singular_body->force_y += force_y;
                    singular_body->force_z += force_z;
                    //printf("Distance: %lf\n", dist);
                }else{

                    //obtain perimeter of octant
                    double perimeter = size_region(arr_tree[i]);
                    //check if well separated
                    //printf("%lf\t%lf\t%lf\n", perimeter/dist, perimeter, dist);
                    if ((perimeter / dist) <= THETA) {
                        // we may treat the nodes in this octant as one and proceed
                        //newtons law of universal gravitation: (G*m1*m2/r^3)*r_(x,y,z)
                        double force_x = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_x - arr_tree[i].com_x);
                        double force_y = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_y - arr_tree[i].com_y);
                        double force_z = (double) ((G * singular_body->mass * arr_tree[i].mass) / ((double)pow(dist, 3)+ 0.3))*(singular_body->com_z - arr_tree[i].com_z);

						if((force_x!=force_x)){
		                  force_x = 1000;
		                  //printf("force_x\n");
		                }
		                if((force_y!=force_y)){
		                  force_y = 1000;
		                  //printf("force_y\n");
		                }
		                if((force_z!=force_z)){
		                  force_z = 1000;
		                  //printf("force_z\n");
		                }

						singular_body->force_x += force_x;
                        singular_body->force_y += force_y;
                        singular_body->force_z += force_z;

                        //skip children
                        //printf("skipped\n");
                        i = arr_tree[i].furthest_right;
                    }
                }
            }
        }
        __syncthreads();
        //TODO: do point update
        //do velocity calc change here
        singular_body->vel_x = (double)(singular_body->vel_x + (singular_body->force_x/singular_body->mass)*delta_t); //(double)fmod((double)(singular_body->vel_x + (singular_body->force_x/singular_body->mass)*delta_t), (double)100.0);
        singular_body->vel_y = (double)(singular_body->vel_y + (singular_body->force_y/singular_body->mass)*delta_t);//(double)fmod((double)(singular_body->vel_y + (singular_body->force_y/singular_body->mass)*delta_t), (double)100.0);
        singular_body->vel_z = (double)(singular_body->vel_z + (singular_body->force_z/singular_body->mass)*delta_t);//(double)fmod((double)(singular_body->vel_z + (singular_body->force_z/singular_body->mass)*delta_t), (double)100.0);
        //update position
		singular_body->com_x = singular_body->com_x + singular_body->vel_x*delta_t;
        singular_body->com_y = singular_body->com_y + singular_body->vel_y*delta_t;
        singular_body->com_z = singular_body->com_z + singular_body->vel_z*delta_t;


		while(abs(singular_body->com_x)>bound){
			double rem = 0;
			if(singular_body->com_x > bound){
				rem = singular_body->com_x - bound;
				singular_body->com_x = (-1)*bound + (double)fmod(rem, (double)(bound-1));
			}else if(singular_body->com_x < (-1)*bound){
				rem = abs(singular_body->com_x)-abs(bound);
				singular_body->com_x = bound - (float)fmod(rem, (double)bound);
			}
			//printf("%lf\n", rem);
			//std::cout<<rem<<std::endl;
		}

		while(abs(singular_body->com_y)>bound){
			double rem = 0;
			if(singular_body->com_y > bound){
				rem = singular_body->com_y - bound;
				singular_body->com_y = (-1)*bound + (double)fmod(rem, (double)(bound-1));
			}else if(singular_body->com_y < (-1)*bound){
				rem = abs(singular_body->com_y)-abs(bound);
				singular_body->com_y = bound - (double)fmod(rem, (double)(bound-1));
			}
			//printf("%lf\n", rem);
			//std::cout<<rem<<std::endl;
		}

		while(abs(singular_body->com_z)>bound){
			double rem = 0;
			if(singular_body->com_z > bound){
				rem = singular_body->com_z - bound;
				singular_body->com_z = (-1)*bound + (double)fmod(rem, (double)(bound-1));
			}else if(singular_body->com_z < (-1)*bound){
				rem = abs(singular_body->com_z)-abs(bound);
				singular_body->com_z = bound - (double)fmod(rem, (double)(bound-1));
			}
			//printf("%lf\n", rem);
			//std::cout<<rem<<std::endl;
		}
    }
}

 /*
  *
  */

int main(int argc, char **argv)
{
	// Open file to export point data, only if setting is set
    ofstream positionFile;
    if (writeOut == 1)
    {
        positionFile.open("../visual/positionFile.txt");
    }
	clock_t start_total = clock(), end_total;
	//first cuda call takes longer so this is to allow for fair comparison by removing that overhead
    float *dData = NULL;
    gpuErrchk(hipMalloc((void **) &dData, 1));
    gpuErrchk(hipFree(dData));

    int i = 0;
    int bound;;//32
    int num_ele;//4096
    int iterations;//128
	if(argc == 4){
		sscanf (argv[1],"%d",&bound);
		sscanf (argv[2],"%d",&num_ele);
		sscanf (argv[3],"%d",&iterations);
	}else{
		bound = 32;
		num_ele = 512;
		iterations = 128;
	}
	
    srand(time(NULL)); //just seed the generator

    vector<body *> *point = new vector<body *>(num_ele);
    //create point array
    for (i = 0; i < num_ele; i++)
    {
        //mass
        double mass = (float)((rand() % 20000) / 1000.0) + 1.0;

        //position
        dim3float pos(
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound,
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound,
            (float)((rand() / (float)RAND_MAX) * 2 * bound) - (float)bound);

        //velocity
        dim3float vel(
            (float)((rand() % 40000) - 20000) / 1000.0,
            (float)((rand() % 40000) - 20000) / 1000.0,
            (float)((rand() % 40000) - 20000) / 1000.0);

        //insert
        body *newBody = create_body(mass, pos, vel);
        (*point)[i] = (newBody);
    }
	// kernel time over all iterations
	float kernel_time = 0;
	float overhead_time = 0;
    //insert into tree and run
    for (int j = 0; j < iterations; j++)
    {	
		
        node *test = malloc_node(-1 * bound, -1 * bound, -1 * bound, bound, bound, bound);
        for (int i = 0; i < num_ele; i++)
        {
            int ele = insert_node(test, (*point)[i], i);
        }
		//overhead timing
		clock_t start, end;
        start = clock();
        //array representation
        int n = get_points(0, test);
        //declare pointers to host and device memory
        arr_node* h_arr_tree;
        arr_node* d_arr_tree;
        //individual body positions in array
        int* h_body_pos;
        int* d_body_pos;
        //allocate host memory
        h_arr_tree = (arr_node*)malloc(n*sizeof(arr_node));
        h_body_pos = (int*)malloc(num_ele*sizeof(int));
        //populate host array
        int pos = 0;
        int garbage = convert_tree_to_array(0, test, h_arr_tree, h_body_pos, &pos);
		//overhead
		end = clock();
		overhead_time += end - start;        
		//free the tree
        free_node(test);


		// overhead
		start = clock();
        //device memory allocation
        hipMalloc((void **)&d_arr_tree, n*sizeof(arr_node));
        hipMalloc((void **)&d_body_pos, num_ele*sizeof(int));

        int nblocks  = num_ele/1024;
		if(!nblocks){
			nblocks++;
		}
        int nthreads = num_ele/nblocks;
        //check if enough threads exist
        if(nthreads*nblocks != num_ele){
            nthreads++;
        }

        //copy relevant data to device
        hipMemcpy(d_arr_tree, h_arr_tree, n*sizeof(arr_node), hipMemcpyHostToDevice);
        hipMemcpy(d_body_pos, h_body_pos, num_ele*sizeof(int), hipMemcpyHostToDevice);
		// overhead
		end = clock();
		overhead_time += end - start;
		
		float time;
		hipEvent_t start_k, stop_k;

		gpuErrchk( hipEventCreate(&start_k) );
		gpuErrchk( hipEventCreate(&stop_k) );
		gpuErrchk( hipEventRecord(start_k, 0) );
        body_update_kernel<<<nblocks,nthreads>>>(n, num_ele, d_arr_tree, d_body_pos, bound);
        //gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipEventRecord(stop_k, 0) );
		gpuErrchk( hipEventSynchronize(stop_k) );
		gpuErrchk( hipEventElapsedTime(&time, start_k, stop_k) );
		//printf("Kernel: %.11f ms\n",sdkGetTimerValue(&timer));
		kernel_time += time;

		// overhead
		start = clock();
        //end of array representation testing
        hipMemcpy(h_arr_tree, d_arr_tree, n*sizeof(arr_node), hipMemcpyDeviceToHost);
        //end of array representation testing

        //insert function updating the bodies
        update_point(h_arr_tree, h_body_pos, point, num_ele);
		// overhead
		end = clock();
		overhead_time += end - start;
        //free memory
        free(h_arr_tree);
        free(h_body_pos);
        hipFree(d_arr_tree);
        hipFree(d_body_pos);

		if (writeOut == 1)
        {
            for (int idx = 0; idx < num_ele; idx++)
            {
                positionFile << (*point)[idx]->com.x << "|" << (*point)[idx]->com.y << "|" << (*point)[idx]->com.x << endl;
            }
            positionFile << endl;
        }
		//cout<<j<<endl;
    }

	for (i = 0; i < num_ele; i++)
    {
        free((*point)[i]);
    }	
	
	end_total = clock();
	kernel_time /= (float)1000;
	printf("Kernel: \t%f\n",kernel_time);
	overhead_time /= (float)CLOCKS_PER_SEC;
	printf("Overhead: %f\n",overhead_time);
	float total_time = end_total - start_total;
	total_time /= (float)CLOCKS_PER_SEC;
	printf("Total: %f\n",total_time);

	// Close the file when no more updates are going to be written
    if (writeOut == 1)
    {
        positionFile.close();
    }

    return 0;
}

//to start pass num_ele = 0
void update_point(arr_node* arr_tree, int* h_body_pos, vector<body *> *point, int num_ele)
{
    for(int i = 0; i < num_ele; i++){
        //get singular body
        arr_node temp = arr_tree[h_body_pos[i]];
        //get vector index
        int idx = temp.body_num;
        //allocate updated values
		//(*point)[idx]->com.cout2();
        (*point)[idx]->mass = temp.mass;
        (*point)[idx]->com.x = temp.com_x;
        (*point)[idx]->com.y = temp.com_y;
        (*point)[idx]->com.z = temp.com_z;
        (*point)[idx]->vel.x = temp.vel_x;
        (*point)[idx]->vel.y = temp.vel_y;
        (*point)[idx]->vel.z = temp.vel_z;
		//if(i == 3028)
			//(*point)[idx]->com.cout2();
    }
}

// get num points in tree

int get_points(int n, node* octree_node){

    n +=1;
    for (int i = 0; i < 8; i++)
    {
        if (octree_node->children[i])
        {
            n = get_points(n, octree_node->children[i]);
        }
    }
    return n;

}

int convert_tree_to_array(int n, node* octree_node, arr_node* arr_tree, int* h_body_pos, int* pos){
    //store relevant data into array from node
    arr_tree[n].body_num = octree_node->body_num;
    arr_tree[n].com_x = octree_node->com.x;
    arr_tree[n].com_y = octree_node->com.y;
    arr_tree[n].com_z = octree_node->com.z;
    arr_tree[n].max_x = octree_node->max.x;
    arr_tree[n].max_y = octree_node->max.y;
    arr_tree[n].max_z = octree_node->max.z;
    arr_tree[n].min_x = octree_node->min.x;
    arr_tree[n].min_y = octree_node->min.y;
    arr_tree[n].min_z = octree_node->min.z;
    arr_tree[n].mass = octree_node->mass;
    arr_tree[n].vel_x = octree_node->vel.x;
    arr_tree[n].vel_y = octree_node->vel.y;
    arr_tree[n].vel_z = octree_node->vel.z;
    if(arr_tree[n].body_num != -1){
        h_body_pos[*pos] = n;
        *pos = *pos + 1;
    }
    int n1 = n;
    n +=1;
    //traverse children recursively
    for (int i = 0; i < 8; i++)
    {
        if (octree_node->children[i])
        {
            n = convert_tree_to_array(n, octree_node->children[i], arr_tree, h_body_pos, pos);
        }
    }
    //store furthest right node for skipping
    arr_tree[n1].furthest_right = n-1;
    return n;
}
